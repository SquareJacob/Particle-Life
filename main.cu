#include "hip/hip_runtime.h"
#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string>
#include <fstream>
#include <windows.h>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
const int WIDTH = 600;
const int HEIGHT = 600;
const int CONTROLWIDTH = 200;

void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

std::string lowercase(std::string str)
{
	std::string result = "";

	for (char ch : str) {
		// Convert each character to lowercase using tolower 
		result += tolower(ch);
	}

	return result;
}

double rmin = 5.0;
double rmax = 50.0;
double repulse = 1.0;
__device__ double force(double attract, double distance, double rmin, double rmax, double repulse) {
	if (distance >= rmax) {
		return 0.0;
	}
	if (distance >= (rmin + rmax) / 2.0) {
		return 2.0 * attract / (rmax - rmin) * (rmax - distance);
	}
	if (distance >= rmin) {
		return 2.0 * attract / (rmax - rmin) * (distance - rmin);
	}
	return repulse * distance / rmin - repulse;
}

double random() {
	return static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
}

const uint8_t numTypes = 6; //KEEP AT 7 OR BELOW
double friction = 1.0;
double speed;
const SDL_Color colors[] = { {255, 0, 0}, {0, 255, 0}, {0, 0, 255}, {255, 80, 237}, {195, 97, 97}, {255, 215, 0} };
class Particle {
public:
	uint8_t type = 0, r = 0, g = 0, b = 0;
	double x = 0.0, y = 0.0, xvel = 0.0, yvel = 0.0;
	void move(double frame) {
		x += frame * xvel;
		y += frame * yvel;
		if (x < 0) {
			x += WIDTH;
		}
		else if (x > WIDTH) {
			x -= WIDTH;
		}
		if (y < 0) {
			y += HEIGHT;
		}
		else if (y > HEIGHT) {
			y -= HEIGHT;
		}
		speed = hypot(xvel, yvel);
		if (speed < friction) {
			xvel = 0.0;
			yvel = 0.0;
		}
		else {
			xvel -= friction * xvel / speed;
			yvel -= friction * yvel / speed;
		}
	}
	void draw() {
		SDL_SetRenderDrawColor(renderer, r, g, b, 255);
		SDL_RenderDrawPoint(renderer, static_cast<int>(x), static_cast<int>(y));
	}
};

class Button {
public:
	uint8_t r = 0, g = 0, b = 0;
	SDL_Rect rect = { 0, 0, 0, 0 };
	bool hovered() {
		return rect.x < mouseX && mouseX < rect.x + rect.w && rect.y < mouseY && mouseY < rect.y + rect.h;
	}
	void draw() {
		SDL_SetRenderDrawColor(renderer, r, g, b, 255);
		SDL_RenderFillRect(renderer, &rect);
		SDL_SetRenderDrawColor(renderer, 255, 255, 255, 255);
		SDL_RenderDrawRect(renderer, &rect);
	}
	void setRGB(double attraction) {
		if (attraction > 0.0) {
			g = 255 * attraction;
			r = 0;
		}
		else {
			r = 255 * -attraction;
			g = 0;
		}
	}
};
Button attractButtons[numTypes * numTypes]; //numTypes * [type1] + [type2], where type1 attracts type2
Button exportButton;

TTF_Font* font;
class Text {
public:
	SDL_Rect rect = { 0, 0, 0, 0 };
	SDL_Texture* texture = NULL;
	~Text() {
		removeTexture();
	}
	void draw() {
		if (SDL_RenderCopy(renderer, texture, NULL, &rect) != 0) {
			debug(__LINE__, __FILE__);
		}
	}
	void removeTexture() {
		if (texture != NULL) {
			SDL_DestroyTexture(texture);
			texture = NULL;
		}
	}
	void createTexture(std::string text, int height) {
		SDL_Surface* tmp = TTF_RenderText_Solid(font, text.c_str(), { 255, 255, 255 });
		if (tmp == NULL) {
			debug(__LINE__, __FILE__);
			return;
		}
		removeTexture();
		texture = SDL_CreateTextureFromSurface(renderer, tmp);
		rect.h = height;
		rect.w = tmp->w * height / tmp->h;
		SDL_FreeSurface(tmp);
		if (texture == NULL) {
			debug(__LINE__, __FILE__);
		}

	}
};
Text repText, attractText, exportText, outputText;

class Slider {
public:
	Text label;
	Button bar, handle;
	Slider() {
		handle.r = 255;
		handle.g = 255;
		handle.b = 255;
		handle.rect.w = 6;

		bar.r = 255;
		bar.g = 255;
		bar.b = 255;
		bar.rect.h = 6;
	}
	void setText(std::string text, int height) {
		label.createTexture(text, height);
		handle.rect.h = label.rect.h;
	}
	void setPos(int x, int y) {
		label.rect.x = x;
		label.rect.y = y;
		handle.rect.x = label.rect.w + 10 + x - handle.rect.w / 2;
		handle.rect.y = y;
		bar.rect.x = x + label.rect.w + 10;
		bar.rect.y = y - (bar.rect.h - handle.rect.h) / 2;
	}
	void setValue(double value) {
		handle.rect.x = bar.rect.x - handle.rect.w / 2 + value * bar.rect.w;
	}
	double getValue() {
		return static_cast<float>(handle.rect.x + handle.rect.w / 2 - bar.rect.x) / static_cast<float>(bar.rect.w);
	}
	double update() {
		if ((bar.hovered() || handle.hovered()) && buttons.contains(1)) {
			handle.rect.x = std::max(bar.rect.x, std::min(mouseX, bar.rect.x + bar.rect.w)) - handle.rect.w / 2;
		}
		bar.draw();
		handle.draw();
		label.draw();
		return getValue();
	}
};
Slider rminSlider, rmaxSlider, repSlider, fricSlider;
SDL_Rect repRect = { 0 };

const uint16_t THREADS = 256; //keep square
const uint16_t numParticles = 6000; //Keep as a multiple of sqrt(THREADS)
const dim3 BLOCKS(375, 375); //numParticles / sqrt(THREADS)

Particle particles[numParticles];
Particle* d_particles;
size_t p_size = sizeof(Particle) * static_cast<size_t>(numParticles);
double attractions[numTypes * numTypes]; //numTypes * [type1] + [type2], where type1 attracts type2
double* d_attractions;

bool exporting = false;
std::string key, fileName;

void allocateAttractions() {
	hipMalloc((void**)&d_attractions, sizeof(double) * static_cast<size_t>(numTypes * numTypes));
	hipMemcpy(d_attractions, attractions, sizeof(double) * static_cast<size_t>(numTypes * numTypes), hipMemcpyHostToDevice);
}

void setAttractions() {
	hipFree(d_attractions);
	for (uint8_t i = 0; i < numTypes * numTypes; i++) {
		attractions[i] = 2.0 * random() - 1.0;
		attractButtons[i].setRGB(attractions[i]);

	}
	allocateAttractions();
}

void setParticles() {
	uint8_t type;
	for (uint16_t i = 0; i < numParticles; i++) {
		type = rand() % numTypes;
		particles[i].type = type;
		particles[i].r = colors[type].r;
		particles[i].g = colors[type].g;
		particles[i].b = colors[type].b;
		particles[i].x = static_cast<float>(WIDTH) * random();
		particles[i].y = static_cast<float>(HEIGHT) * random();
	}
}

void randomize() {
	setAttractions();
	repSlider.setValue(random());
	rminSlider.setValue(random());
	rmaxSlider.setValue(random());
	fricSlider.setValue(random());
	setParticles();
}

Uint32 startTime, totalTime, startCalc, startDraw;

__global__ void totalForce(Particle particles[numParticles], double attractions[numTypes * numTypes], Uint32 totalTime, double rmin, double rmax, double repulse) {
	double disX, disY, dis, attraction;
	Uint32 index = (blockIdx.x * blockDim.x + blockIdx.y) * static_cast<Uint32>(THREADS) + threadIdx.x;
	Uint16 i = index / numParticles;
	Uint16 j = index % numParticles;
	disX = particles[i].x - particles[j].x;
	if (disX > WIDTH / 2) {
		disX -= WIDTH;
	}
	else if (disX < -WIDTH / 2) {
		disX += WIDTH;
	}
	disY = particles[i].y - particles[j].y;
	if (disY > HEIGHT / 2) {
		disY -= HEIGHT / 2;
	}
	else if (disY < -HEIGHT / 2) {
		disY += HEIGHT / 2;
	}
	dis = hypot(disX, disY);
	if (dis > 0.0) {
		attraction = force(attractions[numTypes * particles[i].type + particles[j].type], dis, rmin, rmax, repulse) / dis;
		//std::cout << ' ' << attraction << std::endl;
		particles[j].xvel += disX * attraction;
		particles[j].yvel += disY * attraction;
	}
}

bool timing = false;
int main(int argc, char* argv[]) {
	std::string path0(argv[0]);
	std::string path = path0.substr(0, path0.length() - 18);
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		std::cerr << "Uh oh, looks like your graphics card sucks, dawg. Can't run this. Womp womp" << std::endl;
		return 1;
	}
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH + CONTROLWIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
			return 0;
		}

		srand(time(0));
		setParticles();
		allocateAttractions();
		Button* a;
		int size = CONTROLWIDTH / (numTypes + 1);
		for (uint8_t i = 0; i < numTypes * numTypes; i++) {
			a = &attractButtons[i];
			a->rect = { WIDTH + size / 2 + size * (i % numTypes), size / 2 + size * (i / numTypes), size, size };
		}
		font = TTF_OpenFont((path + "/font.otf").c_str(), CONTROLWIDTH / 10);
		if (font == NULL) {
			debug(__LINE__, __FILE__);
		}


		repSlider.setText("rep", CONTROLWIDTH / 10);
		repSlider.setPos(WIDTH + 5, size * numTypes + size);
		repSlider.bar.rect.w = CONTROLWIDTH + WIDTH - repSlider.bar.rect.x - 5;
		repSlider.setValue(0.5);

		rminSlider.setText("rmin", repSlider.label.rect.h);
		rminSlider.setPos(repSlider.label.rect.x, repSlider.label.rect.y + repSlider.label.rect.h + 10);
		rminSlider.bar.rect.w = CONTROLWIDTH + WIDTH - rminSlider.bar.rect.x - 5;
		rminSlider.setValue(0.5);

		rmaxSlider.setText("rmax", repSlider.label.rect.h);
		rmaxSlider.setPos(repSlider.label.rect.x, rminSlider.label.rect.y + rminSlider.label.rect.h + 10);
		rmaxSlider.bar.rect.w = CONTROLWIDTH + WIDTH - rmaxSlider.bar.rect.x - 5;
		rmaxSlider.setValue(0.5);

		fricSlider.setText("fric", repSlider.label.rect.h);
		fricSlider.setPos(repSlider.label.rect.x, rmaxSlider.label.rect.y + rmaxSlider.label.rect.h + 10);
		fricSlider.bar.rect.w = CONTROLWIDTH + WIDTH - fricSlider.bar.rect.x - 5;
		fricSlider.setValue(0.5);

		repRect = { 0, fricSlider.label.rect.y + fricSlider.label.rect.h + size / 2, size, size };

		repText.createTexture("Column is attracted to row", CONTROLWIDTH / 10);
		repText.rect.x = WIDTH + (CONTROLWIDTH - repText.rect.w) / 2;
		repText.rect.y = repRect.y + repRect.h + size / 2;

		attractText.createTexture("0", CONTROLWIDTH / 10);
		attractText.rect.x = WIDTH + size / 2;
		attractText.rect.y = repText.rect.y + repText.rect.h + size / 2;

		exportText.createTexture("Export", CONTROLWIDTH / 10);
		exportText.rect.x = attractText.rect.x;
		exportText.rect.y = attractText.rect.y + attractText.rect.h + size / 2;

		exportButton.r = 255;
		exportButton.rect = exportText.rect;

		outputText.rect.x = exportButton.rect.x;
		outputText.rect.y = exportButton.rect.y + exportButton.rect.h + size / 2;

		if (argc > 1) {
			std::ifstream myfile(argv[1]);
			if (myfile.is_open()) {
				std::string line;
				double inputs[numTypes * numTypes + 4];
				uint16_t j = 0;
				while (getline(myfile, line, ' ') && j < numTypes * numTypes + 4) {
					inputs[j] = std::stod(line);
					j++;
				}
				myfile.close();
				for (uint16_t i = 0; i < numTypes * numTypes; i++) {
					attractions[i] = inputs[i];
					attractButtons[i].setRGB(attractions[i]);
				}
				repSlider.setValue(inputs[numTypes * numTypes]);
				rminSlider.setValue(inputs[numTypes * numTypes + 1]);
				rmaxSlider.setValue(inputs[numTypes * numTypes + 2]);
				fricSlider.setValue(inputs[numTypes * numTypes + 3]);
				allocateAttractions();
			}
		}


		hipSetDevice(0);
		hipMalloc((void**)&d_particles, p_size);

		//Main loop
		running = true;
		while (running) {
			startTime = SDL_GetTicks();
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}

			if (currentKeys.contains("B")) {
				randomize();
			}

			startCalc = SDL_GetTicks();
			hipMemcpy(d_particles, particles, p_size, hipMemcpyHostToDevice);
			totalForce << <BLOCKS, THREADS >> > (d_particles, d_attractions, totalTime, rmin, rmax, repulse);
			hipDeviceSynchronize();
			hipMemcpy(particles, d_particles, p_size, hipMemcpyDeviceToHost);
			if (timing) {
				std::cout << "calc time: " << SDL_GetTicks() - startCalc;
			}

			startDraw = SDL_GetTicks();
			SDL_SetRenderDrawColor(renderer, 0, 0, 0, 255);
			SDL_RenderClear(renderer);
			for (uint16_t i = 0; i < numParticles; i++) {
				particles[i].move(0.3);
				particles[i].draw();
			}
			Button* a;
			for (uint16_t i = 0; i < numTypes * numTypes; i++) {
				a = &attractButtons[i];
				if (a->hovered()) {
					if (buttons.contains(1)) {
						attractions[i] = std::min(attractions[i] + 0.01, 1.0);
						allocateAttractions();
						a->setRGB(attractions[i]);
					}
					else if (buttons.contains(3)) {
						attractions[i] = std::max(attractions[i] - 0.01, -1.0);
						allocateAttractions();
						a->setRGB(attractions[i]);
					}
					if (keys.contains("0")) {
						attractions[i] = 0;
						a->setRGB(attractions[i]);
						allocateAttractions();
					}
					attractText.createTexture(std::to_string(attractions[i]), attractText.rect.h);
				}
				a->draw();
			}
			repulse = 2.0 * repSlider.update();
			rmin = 10.0 * rminSlider.update();
			rmax = rmin + 15.0 + 75.0 * rmaxSlider.update();
			friction = 2.0 * fricSlider.update() + 2.0;
			repText.draw();
			attractText.draw();
			if (exportButton.hovered() && currentButtons.contains(1)) {
				exporting = true;
				fileName = "";
			}
			if (exporting) {
				if (currentKeys.size() > 0) {
					key = *currentKeys.begin();
					if (key == "Return") {
						if (fileName.length() == 0) {
							fileName = "Preset";
						}
						fileName += ".pal";
						std::ofstream outFile(path + "/Presets/" + fileName);
						if (!outFile.is_open()) {
							std::cout << path + "/Presets/" + fileName << " didn't work :(" << std::endl;
						}
						for (uint8_t i = 0; i < numTypes * numTypes; i++) {
							outFile << attractions[i] << ' ';
						}
						outFile << repSlider.getValue() << ' ';
						outFile << rminSlider.getValue() << ' ';
						outFile << rmaxSlider.getValue() << ' ';
						outFile << fricSlider.getValue() << std::endl;
						outFile.close();
						exporting = false;
					}
					else if (key == "Space") {
						key = " ";
					}
					else {
						key = lowercase(key);
					}
					if (exporting) {
						fileName += key;
						outputText.createTexture(fileName, exportButton.rect.h);
					}
				}
				if (fileName.length() > 0) {
					outputText.draw();
				}
			}
			exportButton.draw();
			exportText.draw();
			for (uint8_t i = 0; i < numTypes; i++) {
				repRect.x = WIDTH + size / 2 + size * i;
				SDL_SetRenderDrawColor(renderer, colors[i].r, colors[i].g, colors[i].b, 255);
				SDL_RenderFillRect(renderer, &repRect);
			}
			SDL_RenderPresent(renderer);
			if (timing) {
				std::cout << " draw time: " << SDL_GetTicks() - startDraw;
			}

			totalTime = SDL_GetTicks() - startTime;
			if (timing) {
				std::cout << " total time: " << totalTime << std::endl;
			}
			//hipError_t err = hipGetLastError();
			//std::cout << "Error: " << hipGetErrorString(err) << std::endl;
			//std::cout << repulse << std::endl;
		}
		
		//Clean up
		hipFree(d_particles);
		hipFree(d_attractions);
		TTF_CloseFont(font);
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}